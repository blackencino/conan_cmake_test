#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


#include <iostream>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#define GLM_FORCE_ALIGNED_GENTYPES
// #define GLM_FORCE_DEFAULT_ALIGNED_GENTYPES
// #define GLM_FORCE_CXX17
#include <glm/glm.hpp>

void do_vecadd_test() {

    std::cout << "CUDACC: " << CUDA_VERSION << std::endl;
    // H has storage for 4 integers
    thrust::host_vector<int> H(4);

    // initialize individual elements
    H[0] = 14;
    H[1] = 20;
    H[2] = 38;
    H[3] = 46;

    // H.size() returns the size of vector H
    std::cout << "H has size " << H.size() << std::endl;

    // print contents of H
    for (int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << std::endl;

    // resize H
    H.resize(2);

    std::cout << "H now has size " << H.size() << std::endl;

    // Copy host_vector H to device_vector D
    thrust::device_vector<int> D = H;

    // elements of D can be modified
    D[0] = 99;
    D[1] = 88;

    // print contents of D
    for (int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // H and D are automatically deleted when the function returns

    thrust::host_vector<glm::vec4> h_positions(4096);
    thrust::device_vector<glm::vec4> d_positions = h_positions; 


}
